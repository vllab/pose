#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "blas.h"
#include "hip/hip_runtime.h"
#include "utils.h"
}

__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= N) return;
    int f = (index/spatial)%filters;
    
    x[index] = (x[index] - mean[f])/(sqrt(variance[f]) + .00001f);
}

__global__ void normalize_delta_kernel(int N, float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= N) return;
    int f = (index/spatial)%filters;
    
    delta[index] = delta[index] * 1./(sqrt(variance[f]) + .00001f) + variance_delta[f] * 2. * (x[index] - mean[f]) / (spatial * batch) + mean_delta[f]/(spatial*batch);
}

extern "C" void normalize_delta_gpu(float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    size_t N = batch*filters*spatial;
    normalize_delta_kernel<<<cuda_gridsize(N), BLOCK>>>(N, x, mean, variance, mean_delta, variance_delta, batch, filters, spatial, delta);
    check_error(hipPeekAtLastError());
}

__global__ void  variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= filters) return;
    int j,k;
    variance_delta[i] = 0;
    for(j = 0; j < batch; ++j){
        for(k = 0; k < spatial; ++k){
            int index = j*filters*spatial + i*spatial + k;
            variance_delta[i] += delta[index]*(x[index] - mean[i]);
        }
    }
    variance_delta[i] *= -.5 * pow(variance[i] + .00001f, (float)(-3./2.));
}

__global__ void spatial_variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *spatial_variance_delta)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= batch*filters) return;
    int f = i%filters;
    int b = i/filters;

    int k;
    spatial_variance_delta[i] = 0;
    for (k = 0; k < spatial; ++k) {
        int index = b*filters*spatial + f*spatial + k;
        spatial_variance_delta[i] += delta[index]*(x[index] - mean[f]);
    }
    spatial_variance_delta[i] *= -.5 * pow(variance[f] + .00001f, (float)(-3./2.));
}

extern "C" void variance_delta_gpu(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    variance_delta_kernel<<<cuda_gridsize(filters), BLOCK>>>(x, delta, mean, variance, batch, filters, spatial, variance_delta);
    check_error(hipPeekAtLastError());
}

__global__ void accumulate_kernel(float *x, int n, int groups, float *sum)
{
    int k;
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= groups) return;
    sum[i] = 0;
    for(k = 0; k < n; ++k){
        sum[i] += x[k*groups + i];
    }
}

extern "C" void fast_variance_delta_gpu(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *spatial_variance_delta, float *variance_delta)
{
    spatial_variance_delta_kernel<<<cuda_gridsize(filters*batch), BLOCK>>>(x, delta, mean, variance, batch, filters, spatial, spatial_variance_delta);
    check_error(hipPeekAtLastError());
    accumulate_kernel<<<cuda_gridsize(filters), BLOCK>>>(spatial_variance_delta, batch, filters, variance_delta);
    check_error(hipPeekAtLastError());
}

__global__ void spatial_mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *spatial_mean_delta)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= batch*filters) return;
    int f = i%filters;
    int b = i/filters;

    int k;
    spatial_mean_delta[i] = 0;
    for (k = 0; k < spatial; ++k) {
        int index = b*filters*spatial + f*spatial + k;
        spatial_mean_delta[i] += delta[index];
    }
    spatial_mean_delta[i] *= (-1./sqrt(variance[f] + .00001f));
}

extern "C" void fast_mean_delta_gpu(float *delta, float *variance, int batch, int filters, int spatial, float *spatial_mean_delta, float *mean_delta)
{
    spatial_mean_delta_kernel<<<cuda_gridsize(filters*batch), BLOCK>>>(delta, variance, batch, filters, spatial, spatial_mean_delta);
    check_error(hipPeekAtLastError());
    accumulate_kernel<<<cuda_gridsize(filters), BLOCK>>>(spatial_mean_delta, batch, filters, mean_delta);
    check_error(hipPeekAtLastError());
}

__global__ void mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= filters) return;
    int j,k;
    mean_delta[i] = 0;
    for (j = 0; j < batch; ++j) {
        for (k = 0; k < spatial; ++k) {
            int index = j*filters*spatial + i*spatial + k;
            mean_delta[i] += delta[index];
        }
    }
    mean_delta[i] *= (-1./sqrt(variance[i] + .00001f));
}

extern "C" void mean_delta_gpu(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    mean_delta_kernel<<<cuda_gridsize(filters), BLOCK>>>(delta, variance, batch, filters, spatial, mean_delta);
    check_error(hipPeekAtLastError());
}

__global__ void  mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    float scale = 1./(batch * spatial);
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= filters) return;
    int j,k;
    mean[i] = 0;
    for(j = 0; j < batch; ++j){
        for(k = 0; k < spatial; ++k){
            int index = j*filters*spatial + i*spatial + k;
            mean[i] += x[index];
        }
    }
    mean[i] *= scale;
}

__global__ void spatial_variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    float scale = 1./(spatial*batch-1);
    int k;
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= batch*filters) return;
    int f = i%filters;
    int b = i/filters;

    variance[i] = 0;
    for(k = 0; k < spatial; ++k){
        int index = b*filters*spatial + f*spatial + k;
        variance[i] += pow((x[index] - mean[f]), 2);
    }
    variance[i] *= scale;
}

__global__ void variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    float scale = 1./(batch * spatial);
    int j,k;
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= filters) return;
    variance[i] = 0;
    for(j = 0; j < batch; ++j){
        for(k = 0; k < spatial; ++k){
            int index = j*filters*spatial + i*spatial + k;
            variance[i] += pow((x[index] - mean[i]), 2);
        }
    }
    variance[i] *= scale;
}

__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX,  float *Y, int OFFY, int INCY)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) Y[OFFY+i*INCY] += ALPHA*X[OFFX+i*INCX];
}

__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) Y[i*INCY] = pow(X[i*INCX], ALPHA);
}

__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) X[i*INCX] = ALPHA;
}

__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) X[i*INCX] *= ALPHA;
}

__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) X[i*INCX] = ALPHA;
}

__global__ void mask_kernel(int n,  float *x, float mask_num, float *mask)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < n && mask[i] == mask_num) x[i] = mask_num;
}

__global__ void copy_kernel(int N,  float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) Y[i*INCY + OFFY] = X[i*INCX + OFFX];
}

__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) Y[i*INCY] *= X[i*INCX];
}

extern "C" void normalize_gpu(float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    size_t N = batch*filters*spatial;
    normalize_kernel<<<cuda_gridsize(N), BLOCK>>>(N, x, mean, variance, batch, filters, spatial);
    check_error(hipPeekAtLastError());
}

extern "C" void mean_gpu(float *x, int batch, int filters, int spatial, float *mean)
{
    mean_kernel<<<cuda_gridsize(filters), BLOCK>>>(x, batch, filters, spatial, mean);
    check_error(hipPeekAtLastError());
}

extern "C" void fast_mean_gpu(float *x, int batch, int filters, int spatial, float *spatial_mean, float *mean)
{
    mean_kernel<<<cuda_gridsize(filters*batch), BLOCK>>>(x, 1, filters*batch, spatial, spatial_mean);
    check_error(hipPeekAtLastError());
    mean_kernel<<<cuda_gridsize(filters), BLOCK>>>(spatial_mean, batch, filters, 1, mean);
    check_error(hipPeekAtLastError());
}

extern "C" void fast_variance_gpu(float *x, float *mean, int batch, int filters, int spatial, float *spatial_variance, float *variance)
{
    spatial_variance_kernel<<<cuda_gridsize(batch*filters), BLOCK>>>(x, mean, batch, filters, spatial, spatial_variance);
    check_error(hipPeekAtLastError());
    accumulate_kernel<<<cuda_gridsize(filters), BLOCK>>>(spatial_variance, batch, filters, variance);
    check_error(hipPeekAtLastError());
}

extern "C" void variance_gpu(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    variance_kernel<<<cuda_gridsize(filters), BLOCK>>>(x, mean, batch, filters, spatial, variance);
    check_error(hipPeekAtLastError());
}

extern "C" void axpy_ongpu(int N, float ALPHA, float * X, int INCX, float * Y, int INCY)
{
    axpy_ongpu_offset(N, ALPHA, X, 0, INCX, Y, 0, INCY);
}

extern "C" void pow_ongpu(int N, float ALPHA, float * X, int INCX, float * Y, int INCY)
{
    pow_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX, Y, INCY);
    check_error(hipPeekAtLastError());
}

extern "C" void axpy_ongpu_offset(int N, float ALPHA, float * X, int OFFX, int INCX, float * Y, int OFFY, int INCY)
{
    axpy_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, OFFX, INCX, Y, OFFY, INCY);
    check_error(hipPeekAtLastError());
}

extern "C" void copy_ongpu(int N, float * X, int INCX, float * Y, int INCY)
{
    copy_ongpu_offset(N, X, 0, INCX, Y, 0, INCY);
}

extern "C" void mul_ongpu(int N, float * X, int INCX, float * Y, int INCY)
{
    mul_kernel<<<cuda_gridsize(N), BLOCK>>>(N, X, INCX, Y, INCY);
    check_error(hipPeekAtLastError());
}

extern "C" void copy_ongpu_offset(int N, float * X, int OFFX, int INCX, float * Y, int OFFY, int INCY)
{
    copy_kernel<<<cuda_gridsize(N), BLOCK>>>(N, X, OFFX, INCX, Y, OFFY, INCY);
    check_error(hipPeekAtLastError());
}

extern "C" void mask_ongpu(int N, float * X, float mask_num, float * mask)
{
    mask_kernel<<<cuda_gridsize(N), BLOCK>>>(N, X, mask_num, mask);
    check_error(hipPeekAtLastError());
}

extern "C" void const_ongpu(int N, float ALPHA, float * X, int INCX)
{
    const_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX);
    check_error(hipPeekAtLastError());
}

extern "C" void scal_ongpu(int N, float ALPHA, float * X, int INCX)
{
    scal_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX);
    check_error(hipPeekAtLastError());
}

extern "C" void fill_ongpu(int N, float ALPHA, float * X, int INCX)
{
    fill_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX);
    check_error(hipPeekAtLastError());
}
